#include "hip/hip_runtime.h"
#include "cuda/MatrixOp.h"

/*
 * kernel function to compute Hadamard product
 */
__global__ void double_had(double* a, double* b, double* c)
{

	c[blockIdx.x] = a[blockIdx.x] * b[blockIdx.x];

}

void dhad(double* a, double* b, double* c, size_t dim)
{

    double_had<<<dim,1>>>(a,b,c);

}

/*
 * kernel function for addition
 */
__global__ void double_add(double* a, double* b, double* c)
{

	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];

}

void dadd(double* a, double* b, double* c, size_t dim)
{

    double_add<<<dim,1>>>(a,b,c);

}

/*
 * kernel function for subtraction
 */
__global__ void double_subtract(double* a, double* b, double* c)
{

	c[blockIdx.x] = a[blockIdx.x] - b[blockIdx.x];

}

void dsub(double* a, double* b, double* c, size_t dim)
{

    double_subtract<<<dim,1>>>(a,b,c);

}
